#include "hip/hip_runtime.h"
/*
 * Code for Fast k-Nearest Neighbour Search via Prioritized DCI
 *
 * This code implements the method described in the Prioritized DCI paper,
 * which can be found at https://arxiv.org/abs/1703.00440
 *
 * This file is a part of the Dynamic Continuous Indexing reference
 * implementation.
 *
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at https://mozilla.org/MPL/2.0/.
 *
 * Copyright (C) 2020    Ke Li, Shichong Peng, Mehran Aghabozorgi
 */

#include <malloc.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <float.h>
#include "dci.h"
#include "util.h"

/* Sorting functions */
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

/* CUDA runtime */
#include <hip/hip_runtime.h>
#include <hipblas.h>

__device__
float compute_dist_device(const float* const vec1, const float* const vec2,
		const int dim) {
	int i;
	float sq_dist = 0.0;
	for (i = 0; i < dim; i++) {
		sq_dist += (vec1[i] - vec2[i]) * (vec1[i] - vec2[i]);
	}
	return sqrt(sq_dist);
}

__device__
static inline float abs_d(float x) {
	return x > 0 ? x : -x;
}

/* Normalize the input projection vectors. Vectors are normalized along each row. */
__global__ void normalize_proj_vecs(float* const proj_vec, const int dim,
		const int num_indices, const int num_heads) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	/* Note: Assumes num_blocks = num_threads */
	int total_indices = num_indices * num_heads;
	int chunk_size = (total_indices + blockDim.x * blockDim.x - 1)
			/ (blockDim.x * blockDim.x);
	int vec_index;
	for (int j = 0; j < chunk_size; ++j) {
		vec_index = i * chunk_size + j;
		if (vec_index < total_indices) {
			float sq_norm = 0.0;
			for (int k = 0; k < dim; ++k) {
				sq_norm += proj_vec[vec_index * dim + k]
						* proj_vec[vec_index * dim + k];
			}
			float norm = sqrtf(sq_norm);
			for (int k = 0; k < dim; ++k) {
				proj_vec[vec_index * dim + k] /= norm;
			}
		}
	}
}

/* Create matrix with proj_vec dim-dimensional normalized gaussian vectors.
 vectors are normalized along each row */
void dci_gen_proj_vec(float* const proj_vec, const int dim,
		const int num_indices, const int num_heads) {
	/* Generate the random indices */
	rng_parallel_device(proj_vec, dim * num_indices * num_heads, GAUSS_RAND);

	/* Normalize */
	int block_size = 32;
	int thread_size = 32;
	normalize_proj_vecs<<<block_size, thread_size>>>(proj_vec, dim,
			num_indices, num_heads);

	/* Synchronize the threads */
	hipDeviceSynchronize();
}

/* Initializes the master DCI data structure.  */
void dci_init(dci* const dci_inst, const int dim, const int num_heads, const int num_comp_indices,
		const int num_simp_indices, const int devId) {

	int num_indices = num_comp_indices * num_simp_indices;

	dci_inst->dim = dim;
	dci_inst->num_heads = num_heads;
	dci_inst->num_comp_indices = num_comp_indices;
	dci_inst->num_simp_indices = num_simp_indices;

	hipMallocManaged((void **) &dci_inst->proj_vec,
			sizeof(float) * dim * num_indices * num_heads);
	dci_gen_proj_vec(dci_inst->proj_vec, dim, num_indices, num_heads);

	// testing: same project vector for every head
	/*
	for (int h = 0; h < num_heads; h++) {
		for (int i = 0; i < dim * num_indices; i++) {
			dci_inst->proj_vec[i + dim * num_indices * h] = dci_inst->proj_vec[i];
		}
	}
	*/

	/* Variables that initialize to default values */
	dci_inst->num_points = 0;
	dci_inst->indices = NULL;
	dci_inst->data = NULL;
	dci_inst->devID = devId;
}

/* Sort index within each head  
	each block work on same portion of index just that different thread within each block work on differnt head. 
	ex. block 0 work on indices 0-100, thread 5 work on indices for head 1 */
__global__ void sort_indices(dci* const dci_inst, const int num_indices, const int num_heads,
		const int num_points, const int points_per_block) {

	// number of thread assign to a single head
	// ex. 10 thread, 2 head, then 5 thread / head
	int blockDim_head = (int) (blockDim.x / num_heads);
	
	int total = num_indices;
	int chunk_size = (total + blockDim_head - 1) / blockDim_head;
	int curr_head = (int) (threadIdx.x / blockDim_head);
	int curr_head_thread = threadIdx.x % blockDim_head;

	int idx;
	int num_points_in_block = min(
			(int) (dci_inst->num_points - blockIdx.x * points_per_block),
			points_per_block);
	for (int j = 0; j < chunk_size; j++) {
		idx = curr_head_thread * chunk_size + j; // position within each head
		if (idx < total) {
			mix_sort(
					&(dci_inst->indices[idx * dci_inst->num_points
							+ blockIdx.x * points_per_block 
							+ num_points * num_indices * curr_head]), // adjust position for different head
					num_points_in_block);
		}
	}
}

/* Copy data in proj_vec to indices */
__global__ void copy_to_indices(dci* const dci_inst, float* const data_proj,
		const int num_indices, const int num_points, const int num_heads) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int n = num_indices * num_points * num_heads;
	int chunk_size = (n + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	int idx;
	for (int j = 0; j < chunk_size; j++) {
		idx = i * chunk_size + j;
		if (idx < n) {
			dci_inst->indices[idx].key = data_proj[idx];
			dci_inst->indices[idx].value = (idx % num_points); // only consider the position in the current head
		}
	}
}

/* Add data to the master DCI data structure.  */
void dci_add(dci* const dci_inst, const int dim, const int num_points, const int num_heads,
		float* const data, const int block_size, const int thread_size) {

	int num_indices = dci_inst->num_comp_indices * dci_inst->num_simp_indices;
	float *data_proj;
	hipMallocManaged((void **) &data_proj,
			sizeof(float) * num_points * num_indices * num_heads);

	assert(dim == dci_inst->dim);
	assert(num_heads == dci_inst->num_heads);
	assert(dci_inst->num_points == 0);

	hipMallocManaged((void **) &dci_inst->data,
			sizeof(float) * num_points * dim * num_heads);
	dci_inst->data = data;
	hipMallocManaged((void **) &dci_inst->indices,
			sizeof(idx_elem) * num_points * num_indices * num_heads);

	dci_inst->num_points = num_points;

	for (int i = 0; i < num_heads; i++) {
		int proj_vec_id = i * dim * num_indices;
		int data_id = i * num_points * dim;
		int data_proj_id = i * num_points * num_indices;
		matmul_device(
			HIPBLAS_OP_N, 
			HIPBLAS_OP_T, 
			num_indices, 
			num_points,
			dci_inst->dim,
			&(dci_inst->proj_vec[proj_vec_id]), 
			&(dci_inst->data[data_id]), 
			&(data_proj[data_proj_id]), 
			dci_inst->devID
		);
	}
	hipDeviceSynchronize();

	/* Add to indices */
	copy_to_indices	<<<block_size, thread_size>>>(dci_inst, data_proj, num_indices, num_points, num_heads);

	/* Synchronize the threads */
	hipDeviceSynchronize();

	int points_per_block = (dci_inst->num_points + block_size - 1) / block_size;
	/* Sort the indices */
	sort_indices<<<block_size, thread_size>>>(dci_inst, num_indices, num_heads, num_points, points_per_block);

	/* Synchronize the threads */
	hipDeviceSynchronize();

	hipFree(data_proj);
}

__device__
void insertion_sort(idx_elem arr[], int n) {
	int i, j;
	idx_elem key;
	for (i = 1; i < n; i++) {
		key = arr[i];
		j = i - 1;
		while (j >= 0 && arr[j].key > key.key) {
			arr[j + 1] = arr[j];
			j = j - 1;
		}
		arr[j + 1] = key;
	}
}

/* Modified quick_sort to use "mix_sort" below. */
__device__
void quick_sort(idx_elem arr[], int n) {
	// arbitrary pivot
	float pivot_key = arr[n / 2].key;
	idx_elem swp;
	int low = 0;
	int high = n - 1;
	while (low < n || high > 0) {
		while (arr[low].key < pivot_key && low < n) {
			low++;
		}
		while (arr[high].key > pivot_key && high > 0) {
			high--;
		}
		if (low <= high) {
			swp = arr[low];
			arr[low] = arr[high];
			arr[high] = swp;
			low++;
			high--;
		} else {
			if (high > 0) {
				mix_sort(arr, high + 1);
			}
			if (low < n - 1) {
				mix_sort(&arr[low], n - low);
			}
			return;
		}
	}
}

/* Sorting algorithm. If the number of data points is fewer than 64, then it does
 Insertion Sort. Otherwise, it uses Quick Sort. The reasoning is that if there are
 too few data points, then Quick Sort's overhead may be too large. */
__device__
void mix_sort(idx_elem arr[], int n) {
	if (n > 64) {
		quick_sort(arr, n);
	} else {
		insertion_sort(arr, n);
	}
}

/* find the next closest query projection
	blockDim_head represent the number of thread assign to each head
 */
__device__
static inline int dci_next_closest_proj(const idx_elem* const idx,
		int* const left_pos, int* const right_pos, const float query_proj,
		const int num_elems, const int blockDim_head) {
	int cur_pos;
	int lower_bound = -blockDim_head;
	int upper_bound = num_elems + blockDim_head - 1;
	if ((*left_pos <= lower_bound) && (*right_pos >= upper_bound)) {
		cur_pos = lower_bound;
	} else if (*left_pos <= lower_bound) {
		cur_pos = *right_pos;
		(*right_pos) += blockDim_head;
	} else if (*right_pos >= upper_bound) {
		cur_pos = *left_pos;
		(*left_pos) -= blockDim_head;
	} else if (idx[min(*right_pos, num_elems - 1)].key - query_proj
			< query_proj - idx[max(*left_pos, 0)].key) {
		cur_pos = *right_pos;
		(*right_pos) += blockDim_head;
	} else {
		cur_pos = *left_pos;
		(*left_pos) -= blockDim_head;
	}
	return cur_pos;
}

/* Returns the index of the element whose key is the largest that is less than the key
	Returns an integer from -1 to num_elems - 1 inclusive
	Could return -1 if all elements are greater or equal to key */
__device__
static inline int dci_search_index(const idx_elem* const idx, const float key,
		const int num_elems) {
	int start_pos, end_pos, cur_pos;

	start_pos = -1;
	end_pos = num_elems - 1;
	cur_pos = (start_pos + end_pos + 2) / 2;

	while (start_pos < end_pos) {
		if (idx[cur_pos].key < key) {
			start_pos = cur_pos;
		} else {
			end_pos = cur_pos - 1;
		}
		cur_pos = (start_pos + end_pos + 2) / 2;
	}

	return start_pos;
}

/* search index
	blockDim_head represent the number of thread assign to each head
 */
__device__ void search_index(const dci* const dci_inst, const float* const query_proj_column, 
		const int num_indices, const int num_heads, 
		int* const left_pos, int* const right_pos, 
		const int points_per_block, const int blockDim_head) {

	int total = num_indices;
	int chunk_size = (total + blockDim_head - 1) / blockDim_head;
	int curr_head = (int) (threadIdx.x / blockDim_head);
	int curr_head_thread = threadIdx.x % blockDim_head;

	int idx;
	for (int j = 0; j < chunk_size; j++) {
		idx = curr_head_thread * chunk_size + j; // idx of indices within each head
		if (idx < total) {
			left_pos[idx + curr_head * num_indices] = dci_search_index(
				&(dci_inst->indices[idx * dci_inst->num_points
						+ blockIdx.x * points_per_block
						+ dci_inst->num_points * num_indices * curr_head]), // adjust index for each head
				query_proj_column[idx + curr_head * num_indices], // adjust index for each head
				min(dci_inst->num_points - blockIdx.x * points_per_block,
						points_per_block)) - blockDim_head + 1;

			right_pos[idx + curr_head * num_indices] = 
				left_pos[idx + curr_head * num_indices] + blockDim_head;
		}
	}
}

/* assign index priority to each index
*/
__device__ void init_index_priority(const dci* const dci_inst,
		const float* const query_proj_column, 
		const int num_indices, const int num_heads, 
		int* const left_pos, int* const right_pos, float* const index_priority,
		int* const cur_pos, const int points_per_block, const int blockDim_head) {

	int total = num_indices;
	int chunk_size = (total + blockDim_head - 1) / blockDim_head;
	int num_points_in_block = min(
			(int) (dci_inst->num_points - blockIdx.x * points_per_block),
			points_per_block);

	int curr_head = (int) (threadIdx.x / blockDim_head);
	int curr_head_thread = threadIdx.x % blockDim_head;
	
	int idx;
	for (int j = 0; j < chunk_size; j++) {
		idx = curr_head_thread * chunk_size + j; // idx of indices within each head
		if (idx < total && num_points_in_block > 0) {
			cur_pos[idx + curr_head * num_indices] = dci_next_closest_proj(
				&(dci_inst->indices[idx * (dci_inst->num_points) // find the current index within each head
					+ blockIdx.x * points_per_block // get the point that is processed by this block
					+ dci_inst->num_points * num_indices * curr_head]), // start of the head / adjust for current head
				&(left_pos[idx + curr_head * num_indices]), 
				&(right_pos[idx + curr_head * num_indices]),
				query_proj_column[idx + curr_head * num_indices],
				num_points_in_block, blockDim_head);

			int position;
			if ((cur_pos[idx + curr_head * num_indices] < 0) && (cur_pos[idx + curr_head * num_indices] > -blockDim_head)) {
				position = 0;
			} else if ((cur_pos[idx + curr_head * num_indices] < (num_points_in_block + blockDim_head - 1))
					&& (cur_pos[idx + curr_head * num_indices] >= num_points_in_block)) {
				position = num_points_in_block - 1;
			} else {
				position = cur_pos[idx + curr_head * num_indices];
			}

			assert(position >= 0); // There should be at least one point in the index
			assert(position < num_points_in_block);
			index_priority[idx + curr_head * num_indices] = abs_d(
					dci_inst->indices[position + idx * (dci_inst->num_points)	// position of index (single head)
						+ blockIdx.x * points_per_block // position within each index
						+ dci_inst->num_points * num_indices * curr_head].key
							- query_proj_column[idx + curr_head * num_indices]);
		}
	}
}

__global__ void init_counts(const dci* const dci_inst, int* counts) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int total = dci_inst->num_comp_indices * dci_inst->num_points;
	total = dci_inst->num_heads * total;
	int chunk_size = (total + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	for (int j = 0; j < chunk_size; j++) {
		int l = i * chunk_size + j;
		if (l < total) {
			counts[l] = 0;
		}
	}
}

__global__ void init_candidate_dists(const dci* const dci_inst,
		float* candidate_dists) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int total = dci_inst->num_points * dci_inst->num_heads;
	int chunk_size = (total + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	for (int j = 0; j < chunk_size; j++) {
		int l = i * chunk_size + j;
		if (l < total) {
			candidate_dists[l] = -2.0;
		}
	}
}

__global__ void init_candidate_indices(const dci* const dci_inst,
		int* candidate_indices) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int total = dci_inst->num_points;
	int chunk_size = (total + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	for (int j = 0; j < chunk_size; j++) {
		int l = i * chunk_size + j;
		if (l < total) {
			candidate_indices[l] = -1;
		}
	}
}

// Blind querying does not compute distances or look at the values of indexed vectors
// For blind querying, top_candidates is not used; all_candidates is used to store candidates in the order of retrieval
__global__
static void dci_query_single_point_by_block(const dci* const dci_inst,
		const int num_neighbours, const int num_queries, 
		const float* const query, const float* const query_proj_column,
		const dci_query_config query_config, float* const d_top_candidates_dist, 
		int* const d_top_candidates_index, int* const all_candidates, 
		int* counts, float* candidate_dists) {

	int j, h;
	float cur_dist;
	int num_indices = dci_inst->num_comp_indices * dci_inst->num_simp_indices;
	int num_heads = dci_inst->num_heads;

	float last_top_candidate_dist = -1.0; // The distance of the k^th closest candidate found so far
	int num_candidates = 0, last_top_candidate = -1;

	int max_possible_num_candidates = min(
			query_config.max_num_candidates,
			query_config.num_outer_iterations);

	int blockDim_head = (int) (blockDim.x / num_heads);
	int curr_head = (int) (threadIdx.x / blockDim_head);

	// for each head there are a number of thread assign to each head, and head_threadIdx is just thread id adjust to head
	int head_threadIdx = threadIdx.x % blockDim_head;

	int points_per_block = (dci_inst->num_points + gridDim.x - 1) / gridDim.x; // default number of data processed by a block
	int num_points_in_block = min(
			(int) (dci_inst->num_points - blockIdx.x * points_per_block), // should not process data beyond the total number of data
			points_per_block);

	if (num_points_in_block > 0) {
		// shared value is an array, each value in the array is correspond to a head
		// the array size is num_heads, thread work on same head has same shared value
		__shared__ int could_break_all, k, m;
		__shared__ float *top_index_priority;
		__shared__ int *top_h;
		__shared__ int *position;
		__shared__ int *i;
		__shared__ bool *could_break; // Bug fix: resolve infinite loop if thread 0 exits first

		__shared__ int* left_pos;
		__shared__ int* right_pos;
		__shared__ int* cur_pos;
		__shared__ float* index_priority;

		// init variables
		if (threadIdx.x == 0) {
			top_index_priority = new float[num_heads];
			top_h = new int[num_heads];
			position = new int[num_heads];
			i = new int[num_heads];
			could_break =new bool[num_heads];

			left_pos = new int[num_indices * num_heads];
			right_pos = new int[num_indices * num_heads];
			cur_pos = new int[num_indices * num_heads];
			index_priority = new float[num_indices * num_heads];

			could_break[curr_head] = false;
			could_break_all = 0;
			k = 0;
		}

		__syncthreads();

		// left_pos and right_pos already account for multi-head
		search_index(
			dci_inst, 
			query_proj_column, 
			num_indices, 
			num_heads,
			left_pos,
			right_pos,
			points_per_block,
			blockDim_head
		);

		__syncthreads();

		init_index_priority(
			dci_inst, 
			query_proj_column, 
			num_indices, 
			num_heads,
			left_pos, 
			right_pos,
			index_priority, 
			cur_pos, 
			points_per_block,
			blockDim_head
		);

		__syncthreads();

		while (k < num_points_in_block * dci_inst->num_simp_indices * blockDim_head) {

			if (threadIdx.x == 0) {
				m = 0;
			}
			__syncthreads();

			// iterate for each complex index (work properly)
			while (m < dci_inst->num_comp_indices) {
				// first thread only
				// For each complex index, we find the simple index that has the lowest
				// index priority, that is cloest to the query point (projection on projection 
				// vector), this simple index will be top_h

				// inner loop one
				if ((threadIdx.x % blockDim_head) == 0) {
					// Get the top priority and data index in priority queue
					top_index_priority[curr_head] = DBL_MAX;
					top_h[curr_head] = -1;
					for (h = 0; h < dci_inst->num_simp_indices; h++) {
						if (index_priority[h + m * dci_inst->num_simp_indices + curr_head * num_indices]
								< top_index_priority[curr_head]) {
							top_index_priority[curr_head] = index_priority[h 
								+ m * dci_inst->num_simp_indices 
								+ curr_head * num_indices];
							top_h[curr_head] = h;
						}
					}
				}

				__syncthreads();

				if (top_h[curr_head] >= 0) {
					if ((threadIdx.x % blockDim_head) == 0) {
						i[curr_head] = top_h[curr_head] + m * dci_inst->num_simp_indices + curr_head * num_indices;
						position[curr_head] = cur_pos[i[curr_head]];
					}
				}

				__syncthreads();

				if (top_h[curr_head] >= 0) {
					int cur_index = position[curr_head] + head_threadIdx;

					if (cur_index >= 0 && cur_index < num_points_in_block) {
						int cur_point = dci_inst->indices[cur_index
								+ dci_inst->num_points * i[curr_head]
								+ blockIdx.x * points_per_block].value; // cur_point is index within the head (i[curr_head] already adjust to head)

						counts[cur_point + dci_inst->num_points * m
							+ dci_inst->num_comp_indices * dci_inst->num_points * curr_head]++;

						if (counts[cur_point + dci_inst->num_points * m
								+ dci_inst->num_comp_indices * dci_inst->num_points * curr_head]
								== dci_inst->num_simp_indices) { 
						
							// add offset to candidate_dists
							if (candidate_dists[cur_point + dci_inst->num_points * curr_head] == -2.0) {
								if (query_config.blind) {
									candidate_dists[cur_point + dci_inst->num_points * curr_head] = -1.0;
									// lock
									all_candidates[num_candidates
											+ blockIdx.x * max_possible_num_candidates
											+ max_possible_num_candidates * gridDim.x * curr_head] =
											cur_point;
									num_candidates++;		
								} else {
									// Compute distance
									cur_dist = compute_dist_device(
											&(dci_inst->data[cur_point * dci_inst->dim
													+ dci_inst->num_points * dci_inst->dim * curr_head]), 
											&(query[dci_inst->dim * num_queries * curr_head]), dci_inst->dim);

									candidate_dists[cur_point + dci_inst->num_points * curr_head] = cur_dist;
									if (num_candidates < num_neighbours) {
										d_top_candidates_dist[blockIdx.x * num_neighbours
												+ head_threadIdx * num_neighbours
												+ num_candidates
												+ gridDim.x * blockDim.x * num_neighbours * curr_head] = cur_dist;
										d_top_candidates_index[blockIdx.x * num_neighbours
												+ head_threadIdx * num_neighbours
												+ num_candidates
												+ gridDim.x * blockDim.x * num_neighbours * curr_head] = cur_point;
										if (cur_dist > last_top_candidate_dist) {
											last_top_candidate_dist = cur_dist;
											last_top_candidate = num_candidates;
										}
									} else if (cur_dist < last_top_candidate_dist) {
										d_top_candidates_dist[blockIdx.x * num_neighbours
												+ head_threadIdx * num_neighbours
												+ last_top_candidate
												+ gridDim.x * blockDim.x * num_neighbours * curr_head] = cur_dist;
										d_top_candidates_index[blockIdx.x * num_neighbours // each block has only certain number of neighbours
												+ head_threadIdx * num_neighbours
												+ last_top_candidate
												+ gridDim.x * blockDim.x * num_neighbours * curr_head] = cur_point; // adjust by head
										last_top_candidate_dist = -1.0;
										// Assuming num_neighbours less than the min(blockDim) = 32
										// no need to run on gpu
										for (j = 0; j < num_neighbours; j++) {
											if (d_top_candidates_dist[blockIdx.x * num_neighbours
													+ head_threadIdx * num_neighbours
													+ j
													+ gridDim.x * blockDim.x * num_neighbours * curr_head]
													> last_top_candidate_dist) {
												last_top_candidate_dist =
														d_top_candidates_dist[blockIdx.x * num_neighbours
																+ head_threadIdx * num_neighbours
																+ j
																+ gridDim.x * blockDim.x * num_neighbours * curr_head];
												last_top_candidate = j;
											}
										}
									}
									num_candidates++;
								}
							} else {
								if (!query_config.blind) {
									cur_dist = candidate_dists[cur_point + dci_inst->num_points * curr_head];
								}
							}
						}
						
					}
				}

				__syncthreads();

				if (top_h[curr_head] >= 0) {
					// use the first thread work on each head to update
					if ((threadIdx.x % blockDim_head) == 0) {

						cur_pos[i[curr_head]] = dci_next_closest_proj(
								&(dci_inst->indices[i[curr_head] * (dci_inst->num_points)
										+ blockIdx.x * points_per_block]),
								&(left_pos[i[curr_head]]), &(right_pos[i[curr_head]]), query_proj_column[i[curr_head]], // need reconsider
								num_points_in_block, blockDim_head);

						if ((cur_pos[i[curr_head]] < 0) && (cur_pos[i[curr_head]] > -blockDim_head)) {
							position[curr_head] = 0;
						} else if ((cur_pos[i[curr_head]]
								< (num_points_in_block + blockDim_head - 1))
								&& (cur_pos[i[curr_head]] >= num_points_in_block)) {
							position[curr_head] = num_points_in_block - 1;
						} else {
							position[curr_head] = cur_pos[i[curr_head]];
						}

						if (position[curr_head] >= 0 && position[curr_head] < num_points_in_block) {
							index_priority[i[curr_head]] = abs_d(
									dci_inst->indices[position[curr_head]
											+ i[curr_head] * (dci_inst->num_points)
											+ blockIdx.x * points_per_block].key
											- query_proj_column[i[curr_head]]);
						} else {
							index_priority[i[curr_head]] = DBL_MAX;
							cur_pos[i[curr_head]] = -blockDim_head;
						}
					}
				}

				if (threadIdx.x == 0) {
					m = m + 1;
				}
				__syncthreads();
			}

			if ((threadIdx.x % blockDim_head) == 0) {
				if (!could_break[curr_head]) {
					if (num_candidates >= num_neighbours) {
						if (k + 1 >= query_config.num_outer_iterations
										* dci_inst->num_simp_indices
								|| num_candidates >= query_config.max_num_candidates) {
							could_break[curr_head] = true;
							could_break_all++;
						}
					}
				}
			}

			if (threadIdx.x == 0) {
				k = k + 1;
			}

			__syncthreads();

			// need to ensure all could_break is could break 
			if (could_break_all == num_heads) {
			    break;
			}
		}

		__syncthreads();

		// free variables
		if (threadIdx.x == 0) {
			free(left_pos);
			free(right_pos);
			free(cur_pos);
			free(index_priority);

			free(top_index_priority); 
			free(top_h);
			free(position);
			free(i);
			free(could_break);
		}
	}
}

__global__ void mix_sort_kernel(idx_elem* const d_top_candidates,
		const int total) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		mix_sort(d_top_candidates, total);
	}
}

__global__ void update_top(const dci* const dci_inst,
		double* const index_priority, int const comp_index, int* top_h,
		int *mutex) {
	double top_h_priority = DBL_MAX;
	//	Shared top priority array
	extern __shared__ double top_priority[];
	//	Shared top priority index in data array
	extern __shared__ double top_index[];

	unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + tid;
	top_priority[tid] = DBL_MAX;
	top_index[tid] = idx % dci_inst->num_simp_indices;

	while (idx < dci_inst->num_simp_indices) {
		double cur_priority = index_priority[comp_index
				* dci_inst->num_simp_indices + idx];
		if (top_priority[tid] > cur_priority) {
			top_priority[tid] = cur_priority;
			top_index[tid] = idx % dci_inst->num_simp_indices;
		}
		idx += gridDim.x * blockDim.x;
	}
	__syncthreads();
	idx = blockIdx.x * blockDim.x + tid;
	// block-wide reduction
	for (unsigned int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
		if (tid < offset && idx < dci_inst->num_simp_indices) {
			double cur_priority = index_priority[comp_index
					* dci_inst->num_simp_indices + tid];
			double compare_priority = index_priority[comp_index
					* dci_inst->num_simp_indices + tid + offset];
			if (cur_priority > compare_priority) {
				top_priority[tid] = compare_priority;
				top_index[tid] = (blockIdx.x * blockDim.x + tid + offset)
						% dci_inst->num_simp_indices;
			}
		}
		__syncthreads();
	}

	// finally, thread 0 writes the result
	if (threadIdx.x == 0) {
		while (atomicCAS(mutex, 0, 1) != 0)
			;  //lock
		if (top_priority[0] < top_h_priority) {
			top_h_priority = top_priority[0];
			*top_h = top_index[0];
		}
		atomicExch(mutex, 0);  //unlock
	}
}

/*
 * Update the top nearest neighbors with distance from the partial results
 */
void get_top_candidates(int* const nearest_neighbours,
		float* const nearest_neighbour_dists,
		float* const d_top_candidates_dist, int* const d_top_candidates_index,
		const int num_neighbours, const int total) {
	thrust::sort_by_key(thrust::device, d_top_candidates_dist,
			d_top_candidates_dist + total, d_top_candidates_index);
	hipMemcpy(nearest_neighbour_dists, d_top_candidates_dist,
			sizeof(float) * num_neighbours, hipMemcpyDeviceToDevice);
	hipMemcpy(nearest_neighbours, d_top_candidates_index,
			sizeof(int) * num_neighbours, hipMemcpyDeviceToDevice);
}

__global__ void init_dist(float* const candidate_map, const int total,
		const float value) {
	int idx, i = blockDim.x * blockIdx.x + threadIdx.x;
	int chunk_size = (total + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	int j;
	// initialize the counters
	for (j = 0; j < chunk_size; j++) {
		idx = i * chunk_size + j;
		if (idx < total) {
			candidate_map[idx] = value;
		}
	}
}

__global__ void init_candidates(idx_elem* const candidate_map, const int total,
		const float value) {
	int idx, i = blockDim.x * blockIdx.x + threadIdx.x;
	int chunk_size = (total + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	int j;
	// initialize the counters
	for (j = 0; j < chunk_size; j++) {
		idx = i * chunk_size + j;
		if (idx < total) {
			candidate_map[idx].key = value;
			candidate_map[idx].value = -1;
		}
	}
}

__global__ void get_blind_candidate_count(idx_elem* const candidate_map,
		int* const d_all_candidates, const int total, 
		const int num_points, const int num_indices, const int num_heads) {
	int curr_head;
	int idx, i = blockDim.x * blockIdx.x + threadIdx.x;
	int chunk_size = (total * num_heads + blockDim.x * gridDim.x - 1)
			/ (blockDim.x * gridDim.x);
	int j;
	// maintain counts as negative numbers for candidate_map.key in order to reuse mix_sort (ascending)
	for (j = 0; j < chunk_size; j++) {
		idx = i * chunk_size + j;
		curr_head = (int) (idx / total);	// which head the given index belong to

		if (idx < total) {
			candidate_map[d_all_candidates[idx]].key--;
			candidate_map[d_all_candidates[idx]].value = 
					d_all_candidates[idx];
		}
	}
}

/*
 * Update the top nearest neighbors from the partial results
 * total = block_size * max_possible_num_candidates
 */
void get_top_blind_candidates(int* const nearest_neighbours,
		int* const d_all_candidates, const int max_possible_num_candidates,
		const int num_points, const int num_indices,
		const int num_neighbours, const int num_queries, const int num_heads,
		const int total) {
	int i, j;
	idx_elem* candidate_map;
	hipMallocManaged((void **) (&candidate_map),
			sizeof(idx_elem) * total * num_heads);
	int block_size = 1024;
	int thread_size = 32;
	init_candidates<<<block_size, thread_size>>>(candidate_map, total * num_heads, 0);
	// synch all blocks
	hipDeviceSynchronize();
	get_blind_candidate_count<<<block_size, thread_size>>>(candidate_map, d_all_candidates, total, num_points, num_indices, num_heads);
	// synch all blocks
	hipDeviceSynchronize();

	for (j = 0; j < num_heads; j++) {
		mix_sort_kernel<<<1, 1>>>(&(candidate_map[max_possible_num_candidates * block_size * j]), total);
		
		for (i = 0; i < max_possible_num_candidates; i++) {
			nearest_neighbours[i + num_neighbours * num_queries * j] = 
				candidate_map[i + num_neighbours * num_queries * j].value;
		}
	}
}

// change the dimension of query project from (head, query, indices) to (query, head, indices)
__global__ void dci_query_proj_3d_permute(float* const query_proj, float* const query_proj_column, 
		const int num_heads, const int num_queries, const int num_indices) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int total = num_heads * num_queries;
	int chunk_size = (total + blockDim.x * gridDim.x - 1) / (blockDim.x * gridDim.x);

	int idx, head, query;
	for (int j = 0; j < chunk_size; j++) {
		idx = i * chunk_size + j;

		if (idx < total) { // prevent overflow
			head = (int) (idx / num_queries);
			query = idx % num_queries;
			for (int k = 0; k < num_indices; k++) {
				query_proj_column[query * num_heads * num_indices + head * num_indices + k] =
					query_proj[head * num_queries * num_indices + query * num_indices + k];
			}
		}
	}
}

// If blind querying is used, nearest_neighbours must be of size num_queries * max_possible_num_candidates; otherwise, it must be of size num_queries * num_neighbours
// nearest_neighbour_dists can be NULL when blind querying is used
void dci_query(dci* const dci_inst, const int dim, const int num_heads, const int num_queries,
		const float* const query, const int num_neighbours,
		const dci_query_config query_config, int* const nearest_neighbours,
		float* const nearest_neighbour_dists, const int block_size,
		const int thread_size) {

	int num_indices = dci_inst->num_comp_indices * dci_inst->num_simp_indices;
	int max_possible_num_candidates = min(query_config.max_num_candidates,
			query_config.num_outer_iterations);

	assert(dim == dci_inst->dim);
	assert(num_neighbours > 0);
	assert(num_neighbours <= dci_inst->num_points);

	// for fixing timeout
	void* dummy;
	hipMalloc(&dummy, 1);

	// calculate query_proj
	int devId = 0;
	float* query_proj;
	float* query_proj_column;

	hipMallocManaged((void **) (&query_proj),
			sizeof(float) * num_indices * num_queries * num_heads);

	hipMallocManaged((void **) (&query_proj_column),
			sizeof(float) * num_indices * num_queries * num_heads);		

	for (int i = 0; i < num_heads; i++) {
		int query_id = i * dci_inst->dim * num_queries;
		int proj_vec_id = i * dci_inst->dim * num_indices;
		int query_proj_id = i * num_indices * num_queries;

		matmul_device(
			HIPBLAS_OP_N, 
			HIPBLAS_OP_T, 
			num_queries, 
			num_indices,
			dci_inst->dim,
			&(query[query_id]), 
			&(dci_inst->proj_vec[proj_vec_id]), 
			&(query_proj[query_proj_id]), 
			devId
		);
	}
	hipDeviceSynchronize();

	dci_query_proj_3d_permute<<<block_size, thread_size>>>(query_proj, query_proj_column, num_heads, num_queries, num_indices);
	hipDeviceSynchronize();

	// copy query config to device pointer
	dci_query_config* d_query_config;
	hipMallocManaged((void **) (&d_query_config),
			sizeof(dci_query_config));
	hipMemcpy(d_query_config, &query_config, sizeof(dci_query_config),
			hipMemcpyHostToDevice);

	// make the raw nearest neighbors
	int* d_all_candidates;
	hipMallocManaged((void **) (&d_all_candidates),
			sizeof(int) * max_possible_num_candidates * block_size * num_heads);

	float* d_top_candidates_dist;
	hipMalloc((void **) (&d_top_candidates_dist),
			sizeof(float) * num_neighbours * block_size * thread_size * num_heads);
	int* d_top_candidates_index;
	hipMalloc((void **) (&d_top_candidates_index),
			sizeof(int) * num_neighbours * block_size * thread_size * num_heads);

	int* counts;
	hipMallocManaged((void **) (&counts),
			sizeof(int) * dci_inst->num_points * dci_inst->num_comp_indices * num_heads);

	float* candidate_dists;
	hipMallocManaged((void **) (&candidate_dists),
			sizeof(float) * dci_inst->num_points * num_heads);

	for (int j = 0; j < num_queries; j++) { 

		// need to refresh the result holder to avoid carry over results
		init_dist<<<block_size, thread_size>>>(d_top_candidates_dist,
				num_neighbours * block_size * thread_size * num_heads, DBL_MAX);

		hipDeviceSynchronize();
		init_counts<<<block_size, thread_size>>>(dci_inst, counts);
		init_candidate_dists<<<block_size, thread_size>>>(dci_inst,
				candidate_dists);

		hipDeviceSynchronize();

		dci_query_single_point_by_block<<<block_size, thread_size>>>(
				dci_inst,
				num_neighbours, 
				num_queries,
				&(query[j * dim]), 
				&(query_proj_column[j * num_indices * num_heads]),
				*d_query_config,
				d_top_candidates_dist, 
				d_top_candidates_index, 
				d_all_candidates,
				counts, 
				candidate_dists
			);

		hipDeviceSynchronize();

		// output
		if (!query_config.blind) {
			for (int h = 0; h < num_heads; h++) {
				get_top_candidates(
						&(nearest_neighbours[j * num_neighbours + num_neighbours * num_queries * h]),
						&(nearest_neighbour_dists[j * num_neighbours + num_neighbours * num_queries * h]),
						&(d_top_candidates_dist[num_neighbours * block_size * thread_size * h]), 
						&(d_top_candidates_index[num_neighbours * block_size * thread_size * h]),
						num_neighbours, 
						block_size * num_neighbours * thread_size
					);
			}
		} else {
			get_top_blind_candidates(
					&(nearest_neighbours[j * max_possible_num_candidates]),
					d_all_candidates, 
					max_possible_num_candidates,
					dci_inst->num_points,
					num_indices,
					num_neighbours,
					num_queries,
					num_heads,
					block_size * max_possible_num_candidates
				);
		}
	}

	// free the allocated memories
	hipFree(query_proj);
	hipFree(query_proj_column);
	hipFree(d_query_config);
	hipFree(d_all_candidates);
	hipFree(d_top_candidates_dist);
	hipFree(d_top_candidates_index);
	hipFree(counts);
	hipFree(candidate_dists);
}


void dci_clear(dci* const dci_inst) {
	if (dci_inst->indices) {
		hipFree(dci_inst->indices);
		dci_inst->indices = NULL;
	}
	dci_inst->data = NULL;
	dci_inst->num_points = 0;
}

void dci_reset(dci* const dci_inst) {
	dci_clear(dci_inst);
	dci_gen_proj_vec(dci_inst->proj_vec, dci_inst->dim,
			dci_inst->num_comp_indices * dci_inst->num_simp_indices, dci_inst->num_heads);
}

void dci_free(const dci* const dci_inst) {
	if (dci_inst->indices) {
		hipFree(dci_inst->indices);
	}
	hipFree(dci_inst->proj_vec);

}

void dci_dump(const dci* const dci_inst) {
	int i, j;
	int num_indices = dci_inst->num_comp_indices * dci_inst->num_simp_indices;
	for (j = 0; j < num_indices; j++) {
		for (i = 0; i < dci_inst->num_points; i++) {
			printf("%f[%d],",
					dci_inst->indices[i + j * (dci_inst->num_points)].key,
					dci_inst->indices[i + j * (dci_inst->num_points)].value);
		}
		printf("\n");
	}
}
